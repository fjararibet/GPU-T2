#include "hip/hip_runtime.h"
#include "gameOfLife/cuda.hpp"
#include <iostream>

__global__ void gameOfLifeKernel(int *In, int *Out, int n, int m) {
  int curr_col = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_row = blockIdx.y * blockDim.y + threadIdx.y;
  if (curr_row < n && curr_col < m) {
    int neighbor_count = 0;
    for (int row = curr_row - 1; row <= curr_row + 1; row++) {
      if (row < 0 || row >= n)
        continue;
      for (int col = curr_col - 1; col <= curr_col + 1; col++) {
        if (col < 0 || col >= m)
          continue;
        if (row == curr_row && col == curr_col)
          continue;
        if (In[row * m + col]) {
          neighbor_count++;
        }
      }
    }

    int new_cell = In[curr_row * m + curr_col] && (neighbor_count == 2 || neighbor_count == 3);
    if (neighbor_count == 3) {
      new_cell = 1;
    }
    Out[curr_row * m + curr_col] = new_cell;
  }
}
__global__ void gameOfLifeKernelLocalMem(int *In, int *Out, int n, int m) {
  // Block and thread indices
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int col = blockIdx.x * blockDim.x + tx;
  int row = blockIdx.y * blockDim.y + ty;

  // Shared memory tile with 1-cell halo
  extern __shared__ int tile[];

  // Shared memory width includes 2 halo cells (1 on each side)
  int shared_width = blockDim.x + 2;
  int shared_x = tx + 1;
  int shared_y = ty + 1;

  // Load current cell into shared memory, and also halo region
  if (row < n && col < m) {
    tile[shared_y * shared_width + shared_x] = In[row * m + col];
  } else {
    tile[shared_y * shared_width + shared_x] = 0;
  }

  // Load halo cells
  if (tx == 0 && col > 0) {
    tile[shared_y * shared_width + 0] = (row < n) ? In[row * m + (col - 1)] : 0;
  }
  if (tx == blockDim.x - 1 && col < m - 1) {
    tile[shared_y * shared_width + (shared_x + 1)] = (row < n) ? In[row * m + (col + 1)] : 0;
  }
  if (ty == 0 && row > 0) {
    tile[0 * shared_width + shared_x] = (col < m) ? In[(row - 1) * m + col] : 0;
  }
  if (ty == blockDim.y - 1 && row < n - 1) {
    tile[(shared_y + 1) * shared_width + shared_x] = (col < m) ? In[(row + 1) * m + col] : 0;
  }

  // Corners
  if (tx == 0 && ty == 0 && col > 0 && row > 0)
    tile[0 * shared_width + 0] = In[(row - 1) * m + (col - 1)];
  if (tx == 0 && ty == blockDim.y - 1 && col > 0 && row < n - 1)
    tile[(shared_y + 1) * shared_width + 0] = In[(row + 1) * m + (col - 1)];
  if (tx == blockDim.x - 1 && ty == 0 && col < m - 1 && row > 0)
    tile[0 * shared_width + (shared_x + 1)] = In[(row - 1) * m + (col + 1)];
  if (tx == blockDim.x - 1 && ty == blockDim.y - 1 && col < m - 1 && row < n - 1)
    tile[(shared_y + 1) * shared_width + (shared_x + 1)] = In[(row + 1) * m + (col + 1)];

  // Synchronize to make sure the tile is fully loaded
  __syncthreads();

  if (row < n && col < m) {
    int neighbor_count = 0;
    for (int dy = -1; dy <= 1; ++dy) {
      for (int dx = -1; dx <= 1; ++dx) {
        if (dx == 0 && dy == 0)
          continue;
        neighbor_count += tile[(shared_y + dy) * shared_width + (shared_x + dx)];
      }
    }

    int current = tile[shared_y * shared_width + shared_x];
    int new_cell = (current && (neighbor_count == 2 || neighbor_count == 3)) || (!current && neighbor_count == 3);
    Out[row * m + col] = new_cell;
  }
}

GameOfLifeCuda::GameOfLifeCuda(std::vector<std::vector<int>> &grid_, int workgroup_x, int workgroup_y, bool local)
    : grid(grid_), workgroup_x(workgroup_x), workgroup_y(workgroup_y), local(local) {
  n = grid.size();
  m = grid[0].size();
  int N_ELEMENTS = n * m;

  hostIn.resize(N_ELEMENTS);
  hostOut.resize(N_ELEMENTS);

  for (size_t i = 0; i < n; ++i)
    for (size_t j = 0; j < m; ++j)
      hostIn[i * m + j] = grid[i][j];

  hipMalloc(&deviceIn, N_ELEMENTS * sizeof(int));
  hipMalloc(&deviceOut, N_ELEMENTS * sizeof(int));

  hipMemcpy(deviceIn, hostIn.data(), N_ELEMENTS * sizeof(int), hipMemcpyHostToDevice);
}

void GameOfLifeCuda::tick() {
  int N_ELEMENTS = n * m;
  int threadsPerBlockX = workgroup_x;
  int threadsPerBlockY = workgroup_y;
  dim3 blockDim(threadsPerBlockX, threadsPerBlockY);
  int TILE_WIDTH = blockDim.x;
  int TILE_HEIGHT = blockDim.y;
  int sharedMemSize = (TILE_WIDTH + 2) * (TILE_HEIGHT + 2) * sizeof(int);

  dim3 gridDim((m + threadsPerBlockX - 1) / threadsPerBlockX, (n + threadsPerBlockY - 1) / threadsPerBlockY);

  if (local) {
    gameOfLifeKernelLocalMem<<<gridDim, blockDim, sharedMemSize>>>(deviceIn, deviceOut, n, m);
  } else {
    gameOfLifeKernel<<<gridDim, blockDim>>>(deviceIn, deviceOut, n, m);
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA launch error: " << hipGetErrorString(err) << std::endl;
  }

  hipDeviceSynchronize();

  err = hipMemcpy(hostOut.data(), deviceOut, N_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
  }

  hipMemcpy(hostOut.data(), deviceOut, N_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < n; ++i)
    for (size_t j = 0; j < m; ++j)
      grid[i][j] = hostOut[i * m + j];

  std::swap(deviceIn, deviceOut);
  std::swap(hostIn, hostOut);

  hipMemcpy(deviceIn, hostIn.data(), N_ELEMENTS * sizeof(int), hipMemcpyHostToDevice);
}

const std::vector<std::vector<int>> GameOfLifeCuda::get_grid() const { return grid; }
